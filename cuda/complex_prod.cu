#include "luaT.h"
#include "THC/THC.h"
#include <hipfft/hipfft.h>
#include "fft_prod2.cu"
#include "fill_hermitian.cu"
#include "modulus.cu"
#include "complexInterp.cu"
#include "bias.cu"
#include "crop.cu"
#include "prod.cu"
#include "graph_pool.cu"

/* Performs the equivalent of the following Torch code:
for s=1,nMinibatch do
    for i = 1,nOutputPlanes do
        for j = 1,nInputPlanes do
            complex.addcmul(input[s][j],kernel[i][j],output[s][i])
        end
    end
end

where input size is  [nMinibatch x nInputPlanes x nRows x nCols x 2]
	  kernel size is [nOutputPlanes x nInputPlanes x nRows x nCols x 2]
	  output size is [nMinibatch x nOutputPlanes x nRows x nCols x 2]

This can be thought of as a matrix multiplication between the input and kernel, 
where each entry to the matrix is a 2D complex matrix and scalar product is replaced by 
pointwise complex product.

Note this operation is used during fprop, updateGradInput and accGradParameters when 
training in Fourier domain.
*/



static int prod_fprop_real(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *weight = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

	luaL_argcheck(L, input->nDimension == 3, 2, "input should be 3D tensor");
	luaL_argcheck(L, output->nDimension == 3, 2, "output should be 3D tensor");
	luaL_argcheck(L, weight->nDimension == 3, 2, "kernel should be 3D tensor");

	long nMinibatch = input->size[0];
	long nOutputMaps = weight->size[0];
	long nInputMaps = weight->size[1];
	long dim = input->size[2];

	// raw pointers
	float *input_data = (float*)THCudaTensor_data(input);
	float *weight_data = (float*)THCudaTensor_data(weight);
	float *output_data = (float*)THCudaTensor_data(output);
	
	spectral_prod(input_data, weight_data, output_data, dim,
				nMinibatch, nInputMaps*dim, nOutputMaps*dim,
				nInputMaps, dim, dim, 
                nOutputMaps, nInputMaps*dim, dim);

	return 0;
}

static int prod_fprop_complex(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *weight = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
    bool conjWeight = lua_toboolean(L,4);

	luaL_argcheck(L, input->nDimension == 5, 2, "input should be 4D complex tensor");
	luaL_argcheck(L, output->nDimension == 5, 2, "output should be 4D complex tensor");
	luaL_argcheck(L, weight->nDimension == 5, 2, "kernel should be 4D complex tensor");

	long nMinibatch = input->size[0];
	long nOutputPlanes = weight->size[0];
	long nInputPlanes = weight->size[1];
	long nRows = input->size[2];
	long nCols = input->size[3];

	// raw pointers
	hipComplex *input_data = (hipComplex*)THCudaTensor_data(input);
	hipComplex *weight_data = (hipComplex*)THCudaTensor_data(weight);
	hipComplex *output_data = (hipComplex*)THCudaTensor_data(output);
	
	fourier_prod(input_data, weight_data, output_data, nRows, nCols,
				nMinibatch, nInputPlanes*nRows*nCols, nOutputPlanes*nRows*nCols,
				nInputPlanes, nRows*nCols, nRows*nCols, 
                nOutputPlanes, nInputPlanes*nRows*nCols, nRows*nCols,conjWeight);

	return 0;
}


static int prod_bprop_real(lua_State *L) {
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *weight = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

	luaL_argcheck(L, gradInput->nDimension == 3, 2, "gradInput should be 3D tensor");
	luaL_argcheck(L, weight->nDimension == 3, 2, "weight should be 3D tensor");
	luaL_argcheck(L, gradOutput->nDimension == 3, 2, "gradOutput should be 3D tensor");

	long nMinibatch = gradInput->size[0];
	long nOutputMaps = weight->size[0];
	long nInputMaps = weight->size[1];
	long dim = gradInput->size[2];

	// raw pointers
	float *gradOutput_data = (float*)THCudaTensor_data(gradOutput);
	float *weight_data = (float*)THCudaTensor_data(weight);
	float *gradInput_data = (float*)THCudaTensor_data(gradInput);
	
	spectral_prod(gradOutput_data, weight_data, gradInput_data, dim,
				nMinibatch, nOutputMaps*dim, nInputMaps*dim,
				nOutputMaps, dim, dim*nInputMaps, 
                nInputMaps, dim, dim);

	return 0;
}


static int prod_bprop_complex(lua_State *L) {
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *weight = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
    bool conjWeight = lua_toboolean(L, 4);

	luaL_argcheck(L, gradInput->nDimension == 5, 2, "gradInput should be 4D complex tensor");
	luaL_argcheck(L, weight->nDimension == 5, 2, "weight should be 4D complex tensor");
	luaL_argcheck(L, gradOutput->nDimension == 5, 2, "gradOutput should be 4D complex tensor");

	long nMinibatch = gradInput->size[0];
	long nOutputPlanes = weight->size[0];
	long nInputPlanes = weight->size[1];
	long nRows = gradInput->size[2];
	long nCols = gradInput->size[3];

	// raw pointers
	hipComplex *gradOutput_data = (hipComplex*)THCudaTensor_data(gradOutput);
	hipComplex *weight_data = (hipComplex*)THCudaTensor_data(weight);
	hipComplex *gradInput_data = (hipComplex*)THCudaTensor_data(gradInput);
	
	fourier_prod(gradOutput_data, weight_data, gradInput_data, nRows, nCols,
				nMinibatch, nOutputPlanes*nRows*nCols, nInputPlanes*nRows*nCols,
				nOutputPlanes, nRows*nCols, nRows*nCols*nInputPlanes, 
                nInputPlanes, nRows*nCols, nRows*nCols,conjWeight);

	return 0;
}


static int prod_accgrad_real(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradWeight = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

	luaL_argcheck(L, input->nDimension == 3, 2, "input should be 3D tensor");
	luaL_argcheck(L, gradOutput->nDimension == 3, 2, "gradOutput should be 3D tensor");
	luaL_argcheck(L, gradWeight->nDimension == 3, 2, "gradWeight should be 3D tensor");

	long nMinibatch = input->size[0];
	long nOutputMaps = gradWeight->size[0];
	long nInputMaps = gradWeight->size[1];
	long dim = input->size[2];

	// raw pointers
	float *input_data = (float*)THCudaTensor_data(input);
	float *gradOutput_data = (float*)THCudaTensor_data(gradOutput);
	float *gradWeight_data = (float*)THCudaTensor_data(gradWeight);
	
	spectral_prod(input_data, gradOutput_data, gradWeight_data, dim,
				nInputMaps, dim, dim,
				nMinibatch, nInputMaps*dim, nOutputMaps*dim, 
                nOutputMaps, dim, nInputMaps*dim);

	return 0;
}


static int prod_accgrad_complex(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradWeight = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
    int conjGradOutput = lua_toboolean(L, 4);

	luaL_argcheck(L, input->nDimension == 5, 2, "input should be 4D complex tensor");
	luaL_argcheck(L, gradOutput->nDimension == 5, 2, "gradOutput should be 4D complex tensor");
	luaL_argcheck(L, gradWeight->nDimension == 5, 2, "gradWeight should be 4D complex tensor");

	long nMinibatch = input->size[0];
	long nOutputPlanes = gradWeight->size[0];
	long nInputPlanes = gradWeight->size[1];
	long nRows = input->size[2];
	long nCols = input->size[3];

	// raw pointers
	hipComplex *input_data = (hipComplex*)THCudaTensor_data(input);
	hipComplex *gradOutput_data = (hipComplex*)THCudaTensor_data(gradOutput);
	hipComplex *gradWeight_data = (hipComplex*)THCudaTensor_data(gradWeight);
	
	fourier_prod(input_data, gradOutput_data, gradWeight_data, nRows, nCols,
				nInputPlanes, nRows*nCols, nRows*nCols,
				nMinibatch, nInputPlanes*nRows*nCols, nOutputPlanes*nRows*nCols, 
                nOutputPlanes, nRows*nCols, nInputPlanes*nRows*nCols,conjGradOutput);

	return 0;
}


static int fill_hermitian(lua_State *L) {
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
  THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");

  luaL_argcheck(L, THCudaTensor_isContiguous(input), 1, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(output),2, "output must be contiguous");
  luaL_argcheck(L, input->nDimension == 5, 2, "input should be 4D complex tensor");
  luaL_argcheck(L, output->nDimension == 5, 2, "output should be 4D complex tensor");
  luaL_argcheck(L, input->size[3] == output->size[3]/2+1, 2, "input must have N/2+1 columns");

  long nMinibatch = input->size[0];
  long nInputPlanes = input->size[1];
  long nRows = output->size[2];
  long nCols = output->size[3];
  hipComplex *input_data = (hipComplex*)THCudaTensor_data(input);
  hipComplex *output_data = (hipComplex*)THCudaTensor_data(output);
   
  fill_hermitian_call(input_data, output_data, nMinibatch*nInputPlanes,nRows,nCols);

  return 0;
}


static const struct luaL_reg cucomplex [] = {
  {"prod_fprop_real", prod_fprop_real},
  {"prod_bprop_real", prod_bprop_real},
  {"prod_accgrad_real", prod_accgrad_real},
  {"prod_fprop_complex", prod_fprop_complex},
  {"prod_bprop_complex", prod_bprop_complex},
  {"prod_accgrad_complex",prod_accgrad_complex},
  {"fill_hermitian",fill_hermitian},
  {"modulus_updateGradInput",modulus_updateGradInput},
  {"complexInterp_interpolate",complexInterp_interpolate},
  {"bias_updateOutput", bias_updateOutput},
  {"bias_accGradParameters", bias_accGradParameters},
  {"crop_zeroborders",crop_zeroborders},
  {"graph_pool_fprop", graph_pool_fprop},
  {"graph_pool_bprop", graph_pool_bprop},
  {NULL, NULL}
};

LUA_EXTERNC int luaopen_cucomplex(lua_State *L) {
	luaL_openlib(L,"cucomplex",cucomplex,0);
	return 1;
}





	
