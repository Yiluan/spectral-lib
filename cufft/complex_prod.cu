#include "luaT.h"
#include "THC/THC.h"
#include <hipfft/hipfft.h>
#include "fft_prod2.cu"


/* Performs the equivalent of the following Torch code:
for s=1,nMinibatch do
    for i = 1,nOutputPlanes do
        for j = 1,nInputPlanes do
            complex.addcmul(input[s][j],kernel[i][j],output[s][i])
        end
    end
end

where input size is  [nMinibatch x nInputPlanes x nRows x nCols x 2]
	  kernel size is [nOutputPlanes x nInputPlanes x nRows x nCols x 2]
	  output size is [nMinibatch x nOutputPlanes x nRows x nCols x 2]

This can be thought of as a matrix multiplication between the input and kernel, 
where each entry to the matrix is a 2D complex matrix and scalar product is replaced by 
pointwise complex product.

Note this operation is used during fprop, updateGradInput and accGradParameters when 
training in Fourier domain.
*/

static int prod_fprop(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *weight = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

	luaL_argcheck(L, input->nDimension == 5, 2, "input should be 4D complex tensor");
	luaL_argcheck(L, output->nDimension == 5, 2, "output should be 4D complex tensor");
	luaL_argcheck(L, weight->nDimension == 5, 2, "kernel should be 4D complex tensor");

	long nMinibatch = input->size[0];
	long nOutputPlanes = weight->size[0];
	long nInputPlanes = weight->size[1];
	long nRows = input->size[2];
	long nCols = input->size[3];

	// raw pointers
	hipComplex *input_data = (hipComplex*)THCudaTensor_data(input);
	hipComplex *weight_data = (hipComplex*)THCudaTensor_data(weight);
	hipComplex *output_data = (hipComplex*)THCudaTensor_data(output);
	
	fourier_prod(input_data, weight_data, output_data, nRows, nCols,
				nMinibatch, nInputPlanes*nRows*nCols, nOutputPlanes*nRows*nCols,
				nInputPlanes, nRows*nCols, nRows*nCols, 
				nOutputPlanes, nInputPlanes*nRows*nCols, nRows*nCols);

	return 0;
}

static int prod_bprop(lua_State *L) {
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *weight = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

	luaL_argcheck(L, gradInput->nDimension == 5, 2, "gradInput should be 4D complex tensor");
	luaL_argcheck(L, weight->nDimension == 5, 2, "weight should be 4D complex tensor");
	luaL_argcheck(L, gradOutput->nDimension == 5, 2, "gradOutput should be 4D complex tensor");

	long nMinibatch = gradInput->size[0];
	long nOutputPlanes = weight->size[0];
	long nInputPlanes = weight->size[1];
	long nRows = gradInput->size[2];
	long nCols = gradInput->size[3];

	// raw pointers
	hipComplex *gradOutput_data = (hipComplex*)THCudaTensor_data(gradOutput);
	hipComplex *weight_data = (hipComplex*)THCudaTensor_data(weight);
	hipComplex *gradInput_data = (hipComplex*)THCudaTensor_data(gradInput);
	
	fourier_prod(gradOutput_data, weight_data, gradInput_data, nRows, nCols,
				nMinibatch, nOutputPlanes*nRows*nCols, nInputPlanes*nRows*nCols,
				nOutputPlanes, nRows*nCols, nRows*nCols*nInputPlanes, 
				nInputPlanes, nRows*nCols, nRows*nCols);

	return 0;
}

static int prod_accgrad(lua_State *L) {
	THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 1, "torch.CudaTensor");	
	THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradWeight = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

	luaL_argcheck(L, input->nDimension == 5, 2, "input should be 4D complex tensor");
	luaL_argcheck(L, gradOutput->nDimension == 5, 2, "gradOutput should be 4D complex tensor");
	luaL_argcheck(L, gradWeight->nDimension == 5, 2, "gradWeight should be 4D complex tensor");

	long nMinibatch = input->size[0];
	long nOutputPlanes = gradWeight->size[0];
	long nInputPlanes = gradWeight->size[1];
	long nRows = input->size[2];
	long nCols = input->size[3];

	// raw pointers
	hipComplex *input_data = (hipComplex*)THCudaTensor_data(input);
	hipComplex *gradOutput_data = (hipComplex*)THCudaTensor_data(gradOutput);
	hipComplex *gradWeight_data = (hipComplex*)THCudaTensor_data(gradWeight);
	
	fourier_prod(input_data, gradOutput_data, gradWeight_data, nRows, nCols,
				nInputPlanes, nRows*nCols, nRows*nCols,
				nMinibatch, nInputPlanes*nRows*nCols, nOutputPlanes*nRows*nCols, 
				nOutputPlanes, nRows*nCols, nInputPlanes*nRows*nCols);

	return 0;
}
static const struct luaL_reg cucomplex [] = {
	{"prod_fprop", prod_fprop},
	{"prod_bprop", prod_bprop},
	{"prod_accgrad",prod_accgrad},
	{NULL, NULL}
};

LUA_EXTERNC int luaopen_cucomplex(lua_State *L) {
	luaL_openlib(L,"cucomplex",cucomplex,0);
	return 1;
}





	
