#include "hip/hip_runtime.h"
#include "luaT.h"
#include "THC/THC.h"
#include "arithmetic.h"

__global__ void batch_crop_kernel(float* input,
                           const int nCropRows, const int nCropCols, 
                                         const int iH, const int iW,
                                         const int nPlanes){
  const int plane = blockIdx.x;
  if (plane >= nPlanes)
    return;

  input += plane * iH * iW;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  if (ty < iH && (ty > iH-nCropRows-1 || ty < nCropRows)) {
    input[ty*iW + tx] = 0;
  }
  if (tx < iW && (tx > iW-nCropCols-1 || tx < nCropCols)) {
    input[ty*iW + tx] = 0;
  }
}


// we are assuming the input is real, not complex
static int crop_zeroborders(lua_State *L) {
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
  const int nCropRows = luaL_checknumber(L,2);
  const int nCropCols = luaL_checknumber(L,3);

  const int dim = input->nDimension;
  const int iH = input->size[dim-2];
  const int iW = input->size[dim-1];

  long nPlanes, nInputPlanes, nOutputPlanes;
  bool resize = false;

  if (dim == 4) {
    resize = true;
    nOutputPlanes = input->size[0];
    nInputPlanes = input->size[1];
    nPlanes = nInputPlanes*nOutputPlanes;
    THCudaTensor_resize3d(input, nPlanes, iH, iW);
  }
  else {
    nPlanes = input->size[0];
  }

  float* input_data = (float*)THCudaTensor_data(input);
  assert(iH == iW);

  dim3 threads(iH,iW);
  dim3 blocks(nPlanes);
  batch_crop_kernel<<<blocks, threads>>>(input_data,
                                         nCropRows, nCropCols, 
                                         iH, iW, nPlanes);

  if (resize) {
    THCudaTensor_resize4d(input, nOutputPlanes, nInputPlanes, iH, iW);
  }

  CUDA_LOOK_FOR_ERROR();
  return 0;
}



