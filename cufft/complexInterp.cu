#include "hip/hip_runtime.h"
#include "luaT.h"
#include "THC/THC.h"
#include "arithmetic.h"
#include "hipfft/hipfft.h"

// Both interpolation kernels and inputs are copied into shared memory. 
// Each thread compute one element of the output. 
__global__ void batch_interpolate_kernel_forward(hipComplex* input, hipComplex* output, 
                                         float* kernelRows, float* kernelCols,
                                         const int iH, const int iW,
                                         const int oH, const int oW,
                                         const int nPlanes){
  extern __shared__ float shared_mem[];
  float* R = (float*)shared_mem;
  float* C = (float*)&R[iH*oH];
  hipComplex* S = (hipComplex*)&C[iW*oW];

  const int plane = blockIdx.x;
  if (plane >= nPlanes)
    return;

  input += plane * iH * iW;
  output += plane * oH * oW;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  
  // copy the interpolation kernels and input to shared memory  
  if (ty < iH) { 
    R[ty*oH + tx] = kernelRows[ty*oH + tx];
    C[ty*oW + tx] = kernelCols[ty*oW + tx];
  }
  __syncthreads();

  if (tx <  iW && ty < iH)
    S[ty*iW + tx] = input[ty*iW + tx];
  __syncthreads();
  
  // compute result
  float real = 0;
  float imag = 0;
 
  for (int i = 0; i < iH; ++i) {
    for (int j = 0; j < iW; ++j) {
      real += S[i*iW + j].x * R[i*oH + ty] * C[j*oW + tx];
      imag += S[i*iW + j].y * R[i*oH + ty] * C[j*oW + tx];
    }
  }
  output[ty*oW + tx].x = real;
  output[ty*oW + tx].y = imag;
}
    

// Similar to above kernel, but in the case that the output is smaller than the input. 
__global__ void batch_interpolate_kernel_backward(hipComplex* input, hipComplex* output, 
                                         float* kernelRows, float* kernelCols,
                                         const int iH, const int iW,
                                         const int oH, const int oW,
                                         const int nPlanes){
  extern __shared__ float shared_mem[];
  float* R = (float*)&shared_mem;
  float* C = (float*)&R[iH*oH];
  hipComplex* S = (hipComplex*)&C[iW*oW];

  const int plane = blockIdx.x;
  if (plane >= nPlanes)
    return;

  input += plane * iH * iW;
  output += plane * oH * oW;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  
  // copy the interpolation kernels and input to shared memory  
  if (ty < oH) { 
    R[ty*oH + tx] = kernelRows[ty*oH + tx];
    C[ty*oW + tx] = kernelCols[ty*oW + tx];
  }
  __syncthreads();

  if (tx <  iW && ty < iH)
    S[ty*iW + tx] = input[ty*iW + tx];
  __syncthreads();
  
  if (tx < oW && ty < oH) {
    // compute
    float real = 0;
    float imag = 0;
 
    for (int i = 0; i < iH; ++i) {
      for (int j = 0; j < iW; ++j) {
        real += S[j*iW + i].x * R[ty*iH + j] * C[tx*iW + i];
        imag += S[j*iW + i].y * R[ty*iH + j] * C[tx*iW + i];
      }
    }
    output[ty*oW + tx].x = real;
    output[ty*oW + tx].y = imag;
  }
}


static int complexInterp_interpolate(lua_State *L) {
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *kernelRows = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *kernelCols = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
  THCudaTensor *buffer = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
  const int dim = input->nDimension;
  const long iH = input->size[dim-3];
  const long iW = input->size[dim-2];
  const long oH = output->size[dim-3];
  const long oW = output->size[dim-2];
  long nPlanes, nInputPlanes, nOutputPlanes;
  bool resize = false;

  if (dim == 5) {
    resize = true;
    nOutputPlanes = input->size[0];
    nInputPlanes = input->size[1];
    nPlanes = nInputPlanes*nOutputPlanes;
    THCudaTensor_resize4d(input, nPlanes, iH, iW, 2);
    THCudaTensor_resize4d(output, nPlanes, oH, oW, 2);
  }
  else {
    nPlanes = input->size[0];
  }
  THCudaTensor_resize4d(buffer, nPlanes, iH, oW, 2);

  hipComplex* input_data = (hipComplex*)THCudaTensor_data(input);
  hipComplex* output_data = (hipComplex*)THCudaTensor_data(output);
  hipComplex* buffer_data = (hipComplex*)THCudaTensor_data(buffer);
  float* kernelRows_data = THCudaTensor_data(kernelRows);
  float* kernelCols_data = THCudaTensor_data(kernelCols);
  
  assert(iH == iW);
  assert(oH == oW);
  if (oH >= iH) { 
    dim3 threads(oH,oW);
    dim3 blocks(nPlanes);
    int size = (iH*oH + iW*oW)*sizeof(float) + iH*iW*sizeof(hipComplex);
    batch_interpolate_kernel_forward<<<blocks,threads, size>>>(input_data, output_data, 
                                                               kernelRows_data, kernelCols_data,
                                                               iH, iW, oH, oW, nPlanes);
  }
  else {
    dim3 threads(iH,iW);
    dim3 blocks(nPlanes);
    int size = (iH*oH + iW*oW)*sizeof(float) + iH*iW*sizeof(hipComplex);
    batch_interpolate_kernel_backward<<<blocks,threads, size>>>(input_data, output_data, 
                                                                kernelRows_data, kernelCols_data,
                                                                iH, iW, oH, oW, nPlanes);
  }
  if (resize) {
    THCudaTensor_resize5d(input, nInputPlanes, nOutputPlanes, iH, iW, 2);
    THCudaTensor_resize5d(output, nInputPlanes, nOutputPlanes, oH, oW, 2);
  }

  CUDA_LOOK_FOR_ERROR();
  return 0;
}


